// std::system includes
#include <memory>
#include <iostream>
#include <stdio.h>

// CUDA-C includes
#include <hip/hip_runtime.h>


//#include <helper_cuda.h>

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	for (int dev = 0; dev < deviceCount; ++dev)
	{
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;



		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: %s \n", dev, deviceProp.name);
		printf("\nMaxThreadsPerBlock: %d \n", deviceProp.maxThreadsPerBlock);
		printf("\nMaxThreadDim (%d,%d,%d)  \n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("\nMaxGridSize (%d,%d,%d)  \n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

	}

	// finish
	// cudaDeviceReset causes the driver to clean up all state. While 
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling cudaDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	char ch;
	std::cin >> ch;

	exit(EXIT_SUCCESS);
}